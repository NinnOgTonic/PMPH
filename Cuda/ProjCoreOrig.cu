#include "hip/hip_runtime.h"
#include "ProjHelperFun.h"
#include "Constants.h"
#include <stdlib.h>
#include <time.h>
#include <sys/time.h>

static __global__ void
updateParams_kernel(const REAL alpha, const REAL beta, const REAL nu, REAL *myVarX, REAL *myVarY, REAL *myX, REAL *myY, int numX, int numY)
{
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;

  if(gidI >= numX || gidJ >= numY)
    return;

  myVarX[gidI * numY + gidJ] = 0.25 * exp(2.0 * (beta  * log(myX[gidI]) + myY[gidJ] + nu));
  myVarY[gidI * numY + gidJ] = 0.25 * exp(2.0 * (alpha * log(myX[gidI]) + myY[gidJ] + nu));

}

static __global__ void
set_payoff_kernel(REAL* myX, REAL* myResult, unsigned int numX, unsigned int numY, unsigned int numO)
{
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;

  if(gidI >= numX || gidJ >= numY || gidO >= numO)
    return;

  REAL strike = 0.001*gidO;
  REAL payoff = MAX(myX[gidI] - strike, (REAL)0.0);
  myResult[(gidO * numY + gidJ) * numX + gidI] = payoff;

}

static __global__ void
rollback_kernel_0(REAL *a, REAL *b, REAL *c, REAL *u, REAL *v, REAL *myResult, REAL *myVarX, REAL *myVarY, REAL *myDxx, REAL *myDyy, REAL dtInv, int numX, int numY)
{
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;
  const unsigned int bidI = blockIdx.x*blockDim.x;
  const unsigned int bidJ = blockIdx.y*blockDim.y;
  const unsigned int lidI = threadIdx.y;
  const unsigned int lidJ = threadIdx.x;
  const unsigned int gidI = bidI + lidI;
  const unsigned int gidJ = bidJ + lidJ;

  extern __shared__ char sh_mem1[];
  REAL *sh_mem   = (REAL*) sh_mem1;
  REAL *localDxx = (REAL*) sh_mem + 34*34;
  REAL *localDyy = (REAL*) localDxx + 32*3;

  if(bidI + lidJ >= numX || bidJ + lidI >= numY) {
    sh_mem[34*(lidJ+1) + lidI + 1] = 0.0;
  } else {
    sh_mem[34*(lidJ+1) + lidI + 1] = myResult[(gidO * numY + bidJ + lidI) * numX + bidI + lidJ];
  }

  if(lidI < 3) {
    if(gidJ < numY) {
      localDyy[lidI * 32 + lidJ] = myDyy[lidI * numY + gidJ];
    }
  } else if(lidI < 6) {
    if(bidI + lidJ < numX) {
      localDxx[(lidI - 3) * 32 + lidJ] = myDxx[(lidI - 3) * numX + bidI + lidJ];
    }
  } else if (lidI == 6) {
    if(bidJ > 0 && bidI + lidJ < numX) {
      sh_mem[34*(lidJ + 1)] = myResult[(gidO * numY + bidJ - 1) * numX + bidI + lidJ];
    } else {
      sh_mem[34*(lidJ + 1)] = 0.0;
    }
  } else if (lidI == 7) {
    if(bidJ + 32 < numY && bidI + lidJ < numX) {
      sh_mem[34*(lidJ + 1) + 33] = myResult[(gidO * numY + bidJ + 32) * numX + bidI + lidJ];
    } else {
      sh_mem[34*(lidJ + 1) + 33] = 0.0;
    }
  } else if (lidI == 8) {
    if(bidJ + lidJ < numY && bidI > 0) {
      sh_mem[lidJ + 1] = myResult[(gidO * numY + bidJ + lidJ) * numX + bidI - 1];
    } else {
      sh_mem[lidJ + 1] = 0.0;
    }
  } else if (lidI == 9) {
    if(bidJ + lidJ < numY && bidI + 32 < numX) {
      sh_mem[34*33 + lidJ + 1] = myResult[(gidO * numY + bidJ + lidJ) * numX + bidI + 32];
    } else {
      sh_mem[34*33 + lidJ + 1] = 0.0;
    }
  }

  __syncthreads();

  if(gidI >= numX || gidJ >= numY) {
    return;
  }

  v[(gidO * numX + gidI) * numY + gidJ] = 2.0 * myVarY[gidI * numY + gidJ] *
    (localDyy[0 * 32 + lidJ] * sh_mem[34*(lidI + 1) + lidJ] +
     localDyy[1 * 32 + lidJ] * sh_mem[34*(lidI + 1) + lidJ + 1] +
     localDyy[2 * 32 + lidJ] * sh_mem[34*(lidI + 1) + lidJ + 2]);

  u[(gidO * numX + gidI) * numY + gidJ] = myVarX[gidI * numY + gidJ] *
    (localDxx[0 * 32 + lidI] * sh_mem[34*lidI + lidJ + 1] +
     localDxx[1 * 32 + lidI] * sh_mem[34*(lidI + 1) + lidJ + 1] +
     localDxx[2 * 32 + lidI] * sh_mem[34*(lidI + 2) + lidJ + 1]) +
    v[(gidO * numX + gidI) * numY + gidJ] +
    dtInv * sh_mem[34*(lidI + 1) + lidJ + 1];

  if(gidO == 0) {
    a[gidI * numY + gidJ]  =       - myVarX[gidI * numY + gidJ] * localDxx[0 * 32 + lidI];
    b[gidI * numY + gidJ]  = dtInv - myVarX[gidI * numY + gidJ] * localDxx[1 * 32 + lidI];
    c[gidI * numY + gidJ]  =       - myVarX[gidI * numY + gidJ] * localDxx[2 * 32 + lidI];
  }
}

static __global__ void
rollback_kernel_1(REAL *a, REAL *c, REAL *yy, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x + 1;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;

  if(gidJ >= numY)
    return;

  yy[gidI * numY + gidJ] = -a[gidI * numY + gidJ] * c[(gidI-1) * numY + gidJ];
}

static __global__ void
rollback_kernel_2(REAL *yy, REAL *b, int numX, int numY) {
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;
  int i;

  if(gidJ >= numY)
    return;

  yy[gidJ] = 1.0 / b[gidJ];

  for(i = 1; i < numX; i++) {
    yy[i * numY + gidJ] = 1.0 / (b[i * numY + gidJ] + yy[i * numY + gidJ] * yy[(i-1) * numY + gidJ]);
  }
}


static __global__ void
rollback_kernel_3(REAL *a, REAL *b, REAL *c, REAL *u, REAL *yy, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;

  if(gidJ >= numY)
    return;

  if(gidO == 0) {
    if(gidI > 0) {
      a[gidI * numY + gidJ] = 1.0 /
        (c[(gidI - 1) * numY + gidJ] *
         yy[(gidI-1) * numY + gidJ] -
         b[gidI * numY + gidJ] /
         a[gidI * numY + gidJ]
         );
    }
    b[gidI * numY + gidJ] = - c[gidI * numY + gidJ] * yy[gidI * numY + gidJ];
  }

  u[(gidO * numX + gidI) * numY + gidJ] = u[(gidO * numX + gidI) * numY + gidJ] * yy[gidI * numY + gidJ];
}

static __global__ void
rollback_kernel_4(REAL *u, REAL *a, REAL *b, int numX, int numY) {
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;
  int i;

  if(gidJ >= numY)
    return;

  for(i = 1; i < numX; i++) {
    u[(gidO * numX + i) * numY + gidJ] += a[i * numY + gidJ] * u[(gidO * numX + i - 1) * numY + gidJ];
  }
  for(i = numX-2; i >= 0; i--) {
    u[(gidO * numX + i) * numY + gidJ] += b[i * numY + gidJ] * u[(gidO * numX + i + 1) * numY + gidJ];
  }
}

static __global__ void
rollback_kernel_5(REAL *a, REAL *b, REAL *c, REAL *y, REAL *u, REAL *v, REAL *myDyy, REAL *myVarY, REAL dtInv, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;

  if(gidJ >= numY)
    return;

  if(gidO == 0) {
    a[gidJ * numX + gidI] =       - myVarY[gidI * numY + gidJ] * myDyy[0 * numY + gidJ];
    b[gidJ * numX + gidI] = dtInv - myVarY[gidI * numY + gidJ] * myDyy[1 * numY + gidJ];
    c[gidJ * numX + gidI] =       - myVarY[gidI * numY + gidJ] * myDyy[2 * numY + gidJ];
  }
  y[(gidO * numX + gidI) * numY + gidJ] = dtInv * u[(gidO * numX + gidI) * numY + gidJ] - 0.5 * v[(gidO * numX + gidI) * numY + gidJ];

}

static __global__ void
rollback_kernel_6(REAL *a, REAL *c, REAL *yy, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidJ = blockIdx.y*blockDim.y + threadIdx.y;

  if(gidJ >= numY)
    return;

  if(gidJ > 0) {
    yy[gidJ * numX + gidI] = -a[gidJ * numX + gidI] * c[(gidJ - 1) * numX + gidI];
  }
}

static __global__ void
rollback_kernel_7(REAL *yy, REAL *b, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  int j;

  if(gidI >= numX)
    return;

  yy[gidI] = 1.0 / b[gidI];

  for(j = 1; j < numY; j++) {
    yy[j * numX + gidI] = 1.0 / (b[j * numX + gidI] + yy[j * numX + gidI] * yy[(j - 1) * numX + gidI]);
  }
}

static __global__ void
rollback_kernel_8(REAL *a, REAL *b, REAL *c, REAL *u, REAL *v, REAL *y, REAL *yy, REAL *myResult, int numX, int numY, REAL dtInv) {
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;
  const unsigned int bidI = blockIdx.x*blockDim.x;
  const unsigned int bidJ = blockIdx.y*blockDim.y;
  const unsigned int lidI = threadIdx.x;
  const unsigned int lidJ = threadIdx.y;
  const unsigned int gidI = bidI + lidI;
  const unsigned int gidJ = bidJ + lidJ;

  if(gidI >= numX)
    return;

  if(gidO == 0) {
    if(gidJ > 0) {
      a[gidJ * numX + gidI] = 1.0 /
        (c[(gidJ - 1) * numX + gidI] *
         yy[(gidJ - 1) * numX + gidI] -
         b[gidJ * numX + gidI] /
         a[gidJ * numX + gidI]);
    }
    b[gidJ * numX + gidI] = -c[gidJ * numX + gidI] * yy[gidJ * numX + gidI];
  }

  myResult[(gidO * numY + gidJ) * numX + gidI] = y[(gidO * numX + gidI) * numY + gidJ] * yy[gidJ * numX + gidI];
}

static __global__ void
rollback_kernel_9(REAL *myResult, REAL *a, REAL *b, int numX, int numY) {
  const unsigned int gidI = blockIdx.x*blockDim.x + threadIdx.x;
  const unsigned int gidO = blockIdx.z*blockDim.z + threadIdx.z;
  int j;

  if(gidI >= numX)
    return;

  for(j = 1; j < numY; j++) {
    myResult[(gidO * numY + j) * numX + gidI] += a[j * numX + gidI] * myResult[(gidO * numY + j - 1) * numX + gidI];
  }
  for(j = numY-2; j >= 0; j--) {
    myResult[(gidO * numY + j) * numX + gidI] += b[j * numX + gidI] * myResult[(gidO * numY + j + 1) * numX + gidI];
  }
}

struct timeval t_start;
long long counters[20];

static void
start()
{
  gettimeofday(&t_start, NULL);
}

static void
end(long long *out)
{

  struct timeval t_end, t_diff;
  gettimeofday(&t_end, NULL);
  timersub(&t_end, &t_start, &t_diff);
  *out += t_diff.tv_sec*1e6+t_diff.tv_usec;
}

static void
rollback(const REAL dtInv, PrivGlobs &globs)
{

  /* v[o][i][j] = myDyy[0..2][j] `dot` myResult[o][j-1..j+1][i] * myVarY[i][j] * 2.0
     u[o][i][j] = myDxx[0..2][i] `dot` myResult[o][j][i-1..i+1] * myVarX[i][j] + v[o][i][j] + dtInv * myResult[o][j][i]
     a[i][j]    =       - myVarX[i][j] * myDxx[0][i]
     b[i][j]    = dtInv - myVarX[i][j] * myDxx[1][i]
     c[i][j]    =       - myVarX[i][j] * myDxx[2][i]
   */
  rollback_kernel_0
    <<<
    dim3(DIVUP(globs.numX, 32), DIVUP(globs.numY, 32), globs.numO),
    dim3(32, 32, 1),
    (34*34 + 32*6)*sizeof(REAL)
    >>>
    (globs.a, globs.b, globs.c, globs.u, globs.v, globs.myResult, globs.myVarX, globs.myVarY, globs.myDxx, globs.myDyy, dtInv, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[0]); start();

  /* yy[i][j] = - a[i][j] * c[i-1][j] */
  rollback_kernel_1
    <<<
    dim3(globs.numX-1, DIVUP(globs.numY, 128), 1),
    dim3(1, 128, 1)
    >>>
    (globs.a, globs.c, globs.yy, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[1]); start();

  /* yy[0][j] = 1.0 / b[0][j]
     yy[i][j] = 1.0 / (b[i][j] + yy[i][j] * yy[i-1][j])*/
  rollback_kernel_2
    <<<
    dim3(1, DIVUP(globs.numY, 128), 1),
    dim3(1, 128, 1)
    >>>
    (globs.yy, globs.b, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[2]); start();

  /* a[i][j]    = 1.0 / (c[i-1][j] * yy[i-1][j] - b[i][j] / a[i][j])
     b[i][j]    = -  c[i][j] * yy[i][j]
     u[o][i][j] = u[o][i][j] * yy[i][j] */
  rollback_kernel_3
    <<<
    dim3(globs.numX, DIVUP(globs.numY, 128), globs.numO),
    dim3(1, 128, 1)
    >>>
    (globs.a, globs.b, globs.c, globs.u, globs.yy, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[3]); start();

  /* u[o][i][j] += a[i][j] * u[o][i-1][j]
     u[o][i][j] += b[i][j] * u[o][i+1][j] */
  rollback_kernel_4
    <<<
    dim3(1, DIVUP(globs.numY, 128), globs.numO),
    dim3(1, 128, 1)
    >>>
    (globs.u, globs.a, globs.b, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[4]); start();

  /* a[j][i] =  =       - 0.25 * myVarY[j][i] * myDyy[0][j]
     b[j][i] =  = dtInv - 0.25 * myVarY[j][i] * myDyy[1][j]
     c[j][i] =  =       - 0.25 * myVarY[j][i] * myDyy[2][j]
     y[o][i][j] = dtInv * u[o][i][j] - 0.5 * v[o][i][j] */
  rollback_kernel_5
    <<<
    dim3(globs.numX, DIVUP(globs.numY, 128), globs.numO),
    dim3(1, 128, 1)
    >>>
    (globs.a, globs.b, globs.c, globs.u, globs.v, globs.y, globs.myDyy, globs.myVarY, dtInv, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[5]); start();

  /* yy[j][i] = -a[j][i] * c[j-1][i] */
  rollback_kernel_6
    <<<
    dim3(DIVUP(globs.numX, 128), globs.numY, 1),
    dim3(128, 1, 1)
    >>>
    (globs.a, globs.c, globs.yy, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[6]); start();

  /* yy[0][i] = 1.0 / b[0][i]
     yy[j][i] = 1.0 / (b[j][i] + yy[j][i] * yy[j-1][i]) */
  rollback_kernel_7
    <<<
    dim3(DIVUP(globs.numX, 128), 1, 1),
    dim3(128, 1, 1)
    >>>
    (globs.yy, globs.b, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[7]); start();

  /* a[j][i] = 1.0 / (c[j-1][i] * yy[j-1][i] - b[j][i] / a[j][i])
     b[j][i] = -c[j][i] * yy[j][i]
     myResult[o][j][i] = y[o][i][j] * yy[j][i] */
  rollback_kernel_8
    <<<
    dim3(DIVUP(globs.numX, 128), globs.numY, globs.numO),
    dim3(128, 1, 1)
    >>>
    (globs.a, globs.b, globs.c, globs.u, globs.v, globs.y, globs.yy, globs.myResult, globs.numX, globs.numY, dtInv);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[8]); start();

  /* myResult[o][j][i] += a[j][i] * myResult[o][j-1][i]
     myResult[o][j][i] += b[j][i] * myResult[o][j+1][i] */
  rollback_kernel_9
    <<<
    dim3(DIVUP(globs.numX, 128), 1, globs.numO),
    dim3(128, 1, 1)
    >>>
    (globs.myResult, globs.a, globs.b, globs.numX, globs.numY);
  checkCudaError(hipGetLastError());
  checkCudaError(hipDeviceSynchronize());
  end(&counters[9]);

}

static void
value(PrivGlobs &globs,
      const REAL s0,
      const REAL t,
      const REAL alpha,
      const REAL nu,
      const REAL beta,
      REAL *res)
{

  set_payoff_kernel
    <<<
    dim3(globs.numX, DIVUP(globs.numY, 128), globs.numO),
    dim3(1, 128, 1)
    >>>
    (globs.myX, globs.myResult, globs.numX, globs.numY, globs.numO);

  for(int i = globs.numT-2; i >= 0; i--) {
    start();

    updateParams_kernel
      <<<
      dim3(globs.numX, DIVUP(globs.numY, 128), 1),
      dim3(1, 128, 1)
      >>>
      (alpha, beta, -0.5 * nu * nu * globs.myTimeline[i], globs.myVarX, globs.myVarY, globs.myX, globs.myY, globs.numX, globs.numY);
    checkCudaError(hipGetLastError());
    checkCudaError(hipDeviceSynchronize());
    end(&counters[10]); start();

    rollback(1.0 / (globs.myTimeline[i+1] - globs.myTimeline[i]), globs);
  }

  for(int i = 0; i < globs.numO; i++) {
    hipMemcpy(&res[i], &globs.myResult[(i * globs.numY + globs.myYindex)*globs.numX + globs.myXindex], sizeof(REAL), hipMemcpyDeviceToHost);
  }
}

void
run_OrigCPU(const unsigned int   outer,
            const unsigned int   numX,
            const unsigned int   numY,
            const unsigned int   numT,
            const REAL           s0,
            const REAL           t,
            const REAL           alpha,
            const REAL           nu,
            const REAL           beta,
            REAL*                res)   // [outer] RESULT
{
  PrivGlobs globs(numX, numY, numT, outer);
  initGrid(s0, alpha, nu, t, numX, numY, numT, outer, globs);
  initOperator(globs.myX, numX, globs.myDxx, outer);
  initOperator(globs.myY, numY, globs.myDyy, outer);

  for(int i = 0; i <= 10; i++) {
    counters[i] = 0;
  }

  value(globs, s0,   t,
        alpha, nu,   beta,
        res);
  for(int i = 0; i <= 10; i++) {
    printf("%lld %d\n", counters[i], i);
  }
}
